
#include <hip/hip_runtime.h>
#ifdef USE_CUDA
#include <cuda.h>
#include <cusparse_v2.h>
#include "../include/sparse.h"

#define CHECK(call)                                                            \
{                                                                              \
    const cudaError_t error = call;                                            \
    if (error != cudaSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                cudaGetErrorString(error));                                    \
    }                                                                          \
}

#define CHECK_CUSPARSE(call)                                                   \
{                                                                              \
    cusparseStatus_t err;                                                      \
    if ((err = (call)) != CUSPARSE_STATUS_SUCCESS)                             \
    {                                                                          \
        fprintf(stderr, "Got error %d at %s:%d\n", err, __FILE__, __LINE__);   \
        cudaError_t cuda_err = cudaGetLastError();                             \
        if (cuda_err != cudaSuccess)                                           \
        {                                                                      \
            fprintf(stderr, "  CUDA error \"%s\" also detected\n",             \
                    cudaGetErrorString(cuda_err));                             \
        }                                                                      \
        exit(1);                                                               \
    }                                                                          \
}

extern "C"
void spmm(float *result, SparseMatrix *sp_matrix, float *matrix, int mm_col) {
    int *a_row, *a_col;
    float *a_val;
    float *b, *c;
    cusparseHandle_t handle;
    cusparseSpMatDescr_t Adescr;
    cusparseDnMatDescr_t Bdescr;
    cusparseDnMatDescr_t Cdescr;

    printf("<<CUDA>>\n");
    CHECK_CUSPARSE(cusparseCreate(&handle));

    CHECK(cudaMalloc((void**)&a_row, sizeof(int)*(sp_matrix->row_size+1)));
    CHECK(cudaMalloc((void**)&a_col, sizeof(int)*(sp_matrix->nnz)));
    CHECK(cudaMalloc((void**)&a_val, sizeof(float)*(sp_matrix->nnz)));
    CHECK(cudaMalloc((void**)&b,     sizeof(float)*(sp_matrix->col_size*mm_col)));
    CHECK(cudaMalloc((void**)&c,     sizeof(float)*(sp_matrix->row_size*mm_col)));

    CHECK(cudaMemcpy(a_row, sp_matrix->row_p, sizeof(int)*(sp_matrix->row_size+1), cudaMemcpyHostToDevice));
    CHECK(cudaMemcpy(a_col, sp_matrix->col_p, sizeof(int)*(sp_matrix->nnz), cudaMemcpyHostToDevice));
    CHECK(cudaMemcpy(a_val, sp_matrix->val,   sizeof(float)*(sp_matrix->nnz), cudaMemcpyHostToDevice));
    CHECK(cudaMemcpy(b,     matrix,           sizeof(float)*(sp_matrix->col_size*mm_col), cudaMemcpyHostToDevice));
    CHECK(cudaMemset(c,     0,                sizeof(float)*(sp_matrix->row_size*mm_col)));

    CHECK_CUSPARSE(
        cusparseCreateCsr(&Adescr, 
            sp_matrix->row_size, sp_matrix->col_size, sp_matrix->nnz, 
            a_row, a_col, a_val, 
            CUSPARSE_INDEX_32I, CUSPARSE_INDEX_32I, CUSPARSE_INDEX_BASE_ZERO, CUDA_R_32F
        )
    );

    CHECK_CUSPARSE(
        cusparseCreateDnMat(&Bdescr, 
            sp_matrix->col_size, mm_col, mm_col, b, 
            CUDA_R_32F, CUSPARSE_ORDER_ROW
        )
    );

    CHECK_CUSPARSE(
        cusparseCreateDnMat(&Cdescr, 
            sp_matrix->row_size, mm_col, mm_col, c, 
            CUDA_R_32F, CUSPARSE_ORDER_ROW
        )
    );

    float alpha = 1;
    float beta = 0;
    size_t buffer_size;
    void *buffer;

    CHECK_CUSPARSE(
        cusparseSpMM_bufferSize(handle, 
        CUSPARSE_OPERATION_NON_TRANSPOSE, CUSPARSE_OPERATION_NON_TRANSPOSE, 
        (void*)&alpha, Adescr, Bdescr, (void*)&beta, Cdescr, 
        CUDA_R_32F, CUSPARSE_SPMM_ALG_DEFAULT, &buffer_size)
    );

    CHECK(cudaMalloc(&buffer, buffer_size));
    CHECK_CUSPARSE(
        cusparseSpMM_preprocess(handle, 
        CUSPARSE_OPERATION_NON_TRANSPOSE, CUSPARSE_OPERATION_NON_TRANSPOSE, 
        (void*)&alpha, Adescr, Bdescr, (void*)&beta, Cdescr, 
        CUDA_R_32F, CUSPARSE_SPMM_ALG_DEFAULT, buffer)
    );

    CHECK_CUSPARSE(
        cusparseSpMM(handle, 
        CUSPARSE_OPERATION_NON_TRANSPOSE, CUSPARSE_OPERATION_NON_TRANSPOSE, 
        (void*)&alpha, Adescr, Bdescr, (void*)&beta, Cdescr, 
        CUDA_R_32F, CUSPARSE_SPMM_ALG_DEFAULT, buffer)
    );
    CHECK(cudaDeviceSynchronize());

    CHECK(cudaMemcpy(result, c, sizeof(float)*(sp_matrix->row_size*mm_col), cudaMemcpyDeviceToHost));
    CHECK(cudaDeviceSynchronize());

    CHECK(cudaFree(a_row));
    CHECK(cudaFree(a_col));
    CHECK(cudaFree(a_val));
    CHECK(cudaFree(b));
    CHECK(cudaFree(c));
    CHECK(cudaFree(buffer));
    
    CHECK_CUSPARSE(cusparseDestroySpMat(Adescr));
    CHECK_CUSPARSE(cusparseDestroyDnMat(Bdescr));
    CHECK_CUSPARSE(cusparseDestroyDnMat(Cdescr));
    CHECK_CUSPARSE(cusparseDestroy(handle));
}

extern "C"
void mm(float *result, float *a, float *b, int row_a, int col_a, int col_b) {
    #ifdef USE_MP
    #pragma omp parallel for
    #endif
    for (int i = 0; i < row_a; i++) {
        for (int j = 0; j < col_a; j++) {
            float sum = 0;
            for (int k = 0; k < col_b; k++) {
                sum += a[i * col_a + k] * b[k * col_b + j];
            }
            result[i * col_b + j] = sum;
        }
    }
}

extern "C"
void relu(float *result, float *a, int size) {
    #ifdef USE_MP
    #pragma omp parallel for
    #endif
    for (int i = 0; i < size; i++) {
        result[i] = (a[i] > 0) ? a[i] : 0;
    }
}

#endif